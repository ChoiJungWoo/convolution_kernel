#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <math.h>
using namespace std;
#define K 3

// declaration of constant memory where the fiter values are stored
__constant__ float cm[K*K];

__device__ void conv(const float* gm,
                        float* convolved,
                        int bh,
                        int bw,
                        int ih,
                        int iw,
                        int ch,
                        int cw,
                        int smH,
                        int smW,
                        int k,
                        float* sm,
                        int gID,
                        int tID,
                        int nT,
                        int rel_row,
                        int rel_col,
                        int nRows,
                        int stopPrefetchRowID,
                        int lastActiveThreadID) {

    for(int i=k; i<=nRows; i++)
    {
        /*
            ----prefetch a pixel value from GM and store it in register----

            all threads fetch the cell value immediately below to the current cell iteratively

            last thread in the block would fetch k cells immediately below the current cell

            boundary check would be needed for the blocks that act on the bottom most partition of the input image to prevent it from prefetching out of image values.
        */
        float reg;
        float *regArr = new float[k];
        if(i <= stopPrefetchRowID){
            reg = gm[i * iw + gID];
            if(tID == lastActiveThreadID){
                for(int j=1; j<=k-1; j++){
                    regArr[j] = gm[i * iw + gID + j];
                }
            }
        }
        // load k * k pixels above the current cell
        float *imgPixels = new float[k*k];
        for(int r=i-k; r<i; r++){
            for(int c=0; c<k; c++){
                /* translate the indices to [0,k] using r - (i-k) as imgPixels is of size k*k */
                imgPixels[(r-i+k)*k + c] = sm[r * smW + tID + c];
            }
        }
        /*multiply image pixel values with filter values (direct convolution) */
        float convolvedCell = 0.0;
        for(int c=0; c<k*k; c++){
            convolvedCell += cm[c]*imgPixels[c];
        }
        //place the convolvedCell value into convolvedMatrix
        int cID = ( ( (rel_row * bh) + (i-k) ) * cw )+( rel_col * nT )+tID;
        convolved[cID] = convolvedCell;
        __syncthreads();
        if(i <= stopPrefetchRowID){
            sm[i * smW + tID] = reg;
            if(tID == lastActiveThreadID){
                for(int j=1; j<=k-1; j++){
                    int sID = i *smW + tID + j;
                    sm[sID] = regArr[j];
                }
            }
        }
        __syncthreads();
    }


}

__global__ void conv_kernel(const float* gm,
                             float* convolved,
                             int bh,
                             int bw,
                             int ih,
                             int iw,
                             int ch,
                             int cw,
                             int smH,
                             int smW,
                             int k) {

    int tID = threadIdx.x;
    int bID = blockIdx.x;
    int nT = blockDim.x;
    int nB = gridDim.x;
    int nBx = iw / nT;
    //printf("num of blocks is %d\n", nB);
    //printf("nB in a row is %d\n", nBx);
    //check for right border or bottom border thread block
    bool isBottomBorder = false;
    bool isRightBorder = false;
    // bottom border thread block
    if(bID >= nB - nBx) {
        //printf("bID : %d is bottom border\n", bID);
        isBottomBorder = true;
    }
    // right border thread block
    if((bID+1) % nBx == 0){
        //printf("bID : %d is right border\n", bID);
        isRightBorder = true;
    }

    // ---------------- Load k rows from GM into SM ----------------------
    extern __shared__ float sm[];
    // rel_row and rel_col maps the Thread Block to appropriate position
    int rel_row = bID / nBx;
    int rel_col = bID % nBx;
    // (rel_row * bh * iw) covers all the cells before row_ids bh, 2bh, 3bh ..
    // gID finally maps threads to cells at rows 0, bh, 2bh, 3bh, ...
    int gID = (rel_row * bh * iw) + (rel_col * nT) + tID;

    for(int i=0; i<k; i++){

        int sID = i * smW + tID;
        sm[sID] = gm[i * iw + gID];
        /* if last thread in the block, it should fetch additional k-1 pixels
           in each row which are needed for computation of the convolution
        */
        if(!isRightBorder && tID == nT-1){
            for(int j=1; j<=k-1; j++){
                sID = i *smW + tID + j;
                sm[sID] = gm[i * iw + gID + j];
            }
        }

    }

    __syncthreads();

    if( !isBottomBorder && !isRightBorder ){
        int lastActiveThreadID = nT - 1;
        int nRows = bh + k - 1;
        int stopPrefetchRowID = nRows;
        conv( gm, convolved, bh, bw,
                ih, iw, ch, cw, smH, smW, k,
                sm, gID, tID, nT, rel_row, rel_col,
                nRows, stopPrefetchRowID, lastActiveThreadID );
    }
    else if( isBottomBorder && isRightBorder ){
        /* make the last k-1 threads in the block to be idle. as there is no convolution needed for them */
        if(tID < (nT - (k-1))){
            int nRows = bh;
            int stopPrefetchRowID = nRows - 1;
            int lastActiveThreadID = nT - k;
            conv( gm, convolved, bh, bw,
                    ih, iw, ch, cw, smH, smW, k,
                    sm, gID, tID, nT, rel_row, rel_col,
                    nRows, stopPrefetchRowID, lastActiveThreadID );
        }
    }
    else if( isBottomBorder ){
        int nRows = bh;
        int stopPrefetchRowID = nRows-1;
        int lastActiveThreadID = nT - 1;
        conv( gm, convolved, bh, bw,
                ih, iw, ch, cw, smH, smW, k,
                sm, gID, tID, nT, rel_row, rel_col,
                nRows, stopPrefetchRowID, lastActiveThreadID );


    }
    else if( isRightBorder ){
        /* make the last k-1 threads in the block to be idle. as there is no convolution needed for them */
        if(tID < (nT - (k-1))){
            int nRows = bh + k - 1;
            int stopPrefetchRowID = nRows;
            int lastActiveThreadID = nT - k;
            conv( gm, convolved, bh, bw,
                    ih, iw, ch, cw, smH, smW, k,
                    sm, gID, tID, nT, rel_row, rel_col,
                    nRows, stopPrefetchRowID, lastActiveThreadID );
        }

    }



}
int main(int argc, char **argv){
    /* set values for image dimensions, block dimensions, filter size, stride ..
       some of the constraints to keep in mind are
        1. the value of k(filter size) should be less than blcH and blcW
        2. stride value(s) should be 1
    */
    int imgH = 20;
    int imgW = 20;
    int blcH = 10;
    int blcW = 10;
    int k    = K;
    int s    = 1;
    int imgDims = imgH * imgW;
    int imgSize = imgDims * sizeof(float);
    // create host array that can hold pixel intensity values
    float *h_img = new float[imgDims];
    for(int i=0; i<imgDims; i++){
        h_img[i] = 1.0;
    }
    // create device array that can hold pixel intensity values in GPU GM
    float *d_img;
    hipMalloc((void **) &d_img, imgSize );
    hipMemcpy(d_img, h_img, imgSize, hipMemcpyHostToDevice);
    // create filter and copy to constant memory
    int filterDims = k * k;
    int filterSize = filterDims * sizeof(float);
    float *filter = new float[filterDims];
    for(int i=0; i<filterDims; i++){
        filter[i] = 0.5;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(cm), filter, filterSize);
    // create host and device array that holds the convoluted matrix
    int convH = ( (imgH - k) / s ) + 1;
    int convW = convH;
    int convDims = convH * convW;
    int convSize = convDims * sizeof(float);
    float *h_convolved = new float[convDims];
    for(int i=0; i<convDims; i++){
        h_convolved[i] = 0;
    }
    float *d_convolved;
    hipMalloc((void **) &d_convolved, convSize);
    hipMemcpy(d_convolved, h_convolved, convSize, hipMemcpyHostToDevice);
    // calculate shared memory size
    int smH = blcH + k - 1;
    int smW = blcW + k - 1;
    int smSize = smH * smW * sizeof(float);
    // call the kernel
    conv_kernel<<<4, 10, smSize>>>(d_img, d_convolved,
                                    blcH, blcW,
                                    imgH, imgW,
                                    convH, convW,
                                    smH, smW,
                                    k);
    hipMemcpy(h_convolved, d_convolved, convSize, hipMemcpyDeviceToHost);
    for(int i=0; i<convH; i++){
        for(int j=0; j<convW; j++){
            cout<<h_convolved[i*convW +j]<<" ";
        }
        cout<<"\n";
    }
    hipDeviceReset();
    delete h_img;
    delete h_convolved;
    return 0;
}
